// # LEETGPU MATRIX TRANSPOSE (Easy)

// # Write a program that transposes a matrix of 32-bit floating point numbers on a GPU. The transpose of a matrix switches its rows and columns. Given a matrix, A, of dimensions (rows, cols), the transpose, A^T, will have dimensions (cols, rows). All matrices are stored in row-major format.

#include <hip/hip_runtime.h>

__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        output[col * rows + row] = input[row * cols + col];
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, float* output, int rows, int cols) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}