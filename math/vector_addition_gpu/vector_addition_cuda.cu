// LEETGPU VECTOR ADDITION

// Implement a program that performs element-wise addition of two vectors containing 32-bit floating point numbers on a GPU. The program should take two input vectors of equal length and produce a single output vector containing their sum.

#include <hip/hip_runtime.h>

__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    // global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // addition within bounds
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vector_add<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize();
}