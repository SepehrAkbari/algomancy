// LEETGPU MATRIX MULTIPLICATION (Easy)

// Write a program that multiplies two matrices of 32-bit floating point numbers on a GPU. Given matrix A of dimensions M x N and matrix B of dimensions N x K, compute the product matrix C = A x B, which will have dimensions M x K. All matrices are stored in row-major format.

#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    // row and col indices for C
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // bound check
    if (!(row < M && col < K)) {
        return;
    }
    else {
        float val = 0.0f;
        // dot product of row of A and column of B
        for (int i = 0; i < N; ++i) {
            // A is M x N and B is N x K
            val += A[row * N + i] * B[i * K + col];
        }
        C[row * K + col] = val;
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
